#include "hip/hip_runtime.h"
#include "mandelbrot.h"

#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line) {
	if (code != hipSuccess) {
		fprintf(stderr,"CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
		exit(code);
	}
}

// setiap thread GPU
__device__ double mandelbrot_gpu(double cr, double ci) {
	double zr = 0.0, zi = 0.0;
	int iter = 0;

	// interior (Cardioid & Period-2 bulb)
	double ci2 = ci * ci;
	double q = (cr - 0.25) * (cr - 0.25) + ci2;
	if (q * (q + (cr - 0.25)) <= 0.25 * ci2 ||
	(cr + 1.0) * (cr + 1.0) + ci2 <= 0.0625) {
		return (double)MAX_ITER;
	}

	double zr2 = 0.0, zi2 = 0.0;
	while (iter < MAX_ITER) {
	zr2 = zr * zr;
	zi2 = zi * zi;

	if (zr2 + zi2 > 4.0) {
		double log_zn = log(zr2 + zi2) / 2.0;
		double nu = log(log_zn / log(2.0)) / log(2.0);
		return (double)iter + 1.0 - nu;
	}

	zi = 2.0 * zr * zi + ci;
	zr = zr2 - zi2 + cr;
	++iter;
	}

	return (double)MAX_ITER;
}

// Kernel utama
__global__ void mandelbrot_kernel(unsigned char* image) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < WIDTH && y < HEIGHT) {
		double cr = X_MIN + (X_MAX - X_MIN) * x / (WIDTH - 1);
		double ci = Y_MIN + (Y_MAX - Y_MIN) * y / (HEIGHT - 1);

		double iter = mandelbrot_gpu(cr, ci);

		int idx = CHANNELS * (y * WIDTH + x);

		double t = 0.1 * iter;
		double t2 = t * t;
		double t3 = t2 * t;
		double one_minus_t = 1.0 - t;
		double one_minus_t2 = one_minus_t * one_minus_t;
		double one_minus_t3 = one_minus_t2 * one_minus_t;

		image[idx + 0] = (unsigned char)((iter >= MAX_ITER) ? 0 : (9.0 * one_minus_t * t3 * 255.0));
		image[idx + 1] = (unsigned char)((iter >= MAX_ITER) ? 0 : (15.0 * one_minus_t2 * t2 * 255.0));
		image[idx + 2] = (unsigned char)((iter >= MAX_ITER) ? 0 : (8.5 * one_minus_t3 * t * 255.0));
	}
}

extern "C" std::chrono::milliseconds mandelbrot_cuda(unsigned char* h_image) {
  const int image_size = WIDTH * HEIGHT * CHANNELS * sizeof(unsigned char);
	
	unsigned char* d_image;
	hipMalloc(&d_image, image_size);

	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks((WIDTH + 15) / 16, (HEIGHT + 15) / 16);

	auto start_time = std::chrono::high_resolution_clock::now();

	CUDA_CHECK(hipMalloc(&d_image, image_size));
	mandelbrot_kernel<<<numBlocks, threadsPerBlock>>>(d_image);
	CUDA_CHECK(hipGetLastError());
	CUDA_CHECK(hipDeviceSynchronize());
	CUDA_CHECK(hipMemcpy(h_image, d_image, image_size, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipFree(d_image));

	hipDeviceSynchronize();

	auto end_time = std::chrono::high_resolution_clock::now();
	auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);

	hipMemcpy(h_image, d_image, image_size, hipMemcpyDeviceToHost);
	
	hipFree(d_image);
	return duration;
}